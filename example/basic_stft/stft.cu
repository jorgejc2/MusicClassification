#include "hip/hip_runtime.h"
#include "wavreader/wavread.h"
#include "dsp/dsp.h"
#include <math.h>
#include <vector>
#include <NumCpp.hpp>
using namespace std;

int main(int argc, char* argv[])
{

    int sample_rate = 4000;
    int length_ts_sec = 3;
    int length_ts1_sec = 1;
    int length_ts2_sec = 3;
    int total_ts_length = length_ts_sec + length_ts1_sec + length_ts2_sec;
    int nc_ts_size = sample_rate * 7; // want 7 seconds worth of samples

    int freq1 = 697;
    int freq2 = 1209;
    int freq3 = 1336;

    /* now an attempt to create a linspace vector with NumCpp */
    nc::NdArray<float> lin_freq1 = nc::linspace<float>(0, M_PI*2*freq1, sample_rate);
    nc::NdArray<float> lin_freq2 = nc::linspace<float>(0, M_PI*2*freq2, sample_rate);
    nc::NdArray<float> lin_freq3 = nc::linspace<float>(0, M_PI*2*freq3, sample_rate);

    vector<float>nc_ts(nc_ts_size, 0.0);
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < sample_rate; j++) {
            nc_ts[i*sample_rate + j] = sin(lin_freq1[j]) + sin(lin_freq2[j]);
            nc_ts[i*sample_rate + j + 4000] = sin(lin_freq1[j]) + sin(lin_freq3[j]);
        }
    }

    for (int i = 0; i < 10; i++)
        printf("%.15f ", nc_ts[i]);
    
    /* if results above match that of Python, then move on to conducting a serial stft */
    // dsp::create_spectogram(&nc_ts, 256, -1);
    // int test_out = dsp::test_cuda();

    int fft_size = 8;

    hipDoubleComplex* freqs = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * fft_size);
    float* cuda_samples = (float*)malloc(sizeof(float) * fft_size);;
    if (freqs == nullptr || cuda_samples == nullptr) {
        printf("failed allocatin \n");
        return 1;
    }

    for (int i = 0; i < fft_size; i++)
        cuda_samples[i] = i;

    dsp::FFT_Setup(cuda_samples, freqs, fft_size);

    for (int i = 0; i < 8; i++)
        printf("%f + i%f\n", freqs[i].x, freqs[i].y);
        // printf("%.3f ", freqs[i]);

    // delete [] freqs;
    // delete [] cuda_samples;
    free(freqs);
    free(cuda_samples);


    return 0;
}
