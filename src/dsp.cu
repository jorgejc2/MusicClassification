#include "hip/hip_runtime.h"
#include "dsp/dsp.h"
// #include <pybind11/pybind11.h>
// #include <pybind11/stl.h>
// #include <pybind11/numpy.h>
#include <iostream>

// namespace py = pybind11;

/* checks for CUDA errors */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* checks if memory could not be allocated */
#define mallocErrchk(ans) { mallocAssert((ans), __FILE__, __LINE__); }
inline void mallocAssert(void* pointer, const char *file, int line, bool abort=true) {
    if (pointer == nullptr)
    {
        fprintf(stderr, "mallocAssert: Returns nullptr at %s %d\n", file, line);
        if (abort) exit(1);
    }
}

__host__ int dsp::create_spectogram(vector<float> *ts, int NFFT = 256, int noverlap = -1) {
    if (noverlap < 0)
        noverlap = NFFT / 2;

    int32_t ts_size = (int32_t)ts->size();

    nc::NdArray<int> starts_original = nc::arange<int>(0, ts_size, NFFT - noverlap);
    nc::NdArray<int> starts = starts_original[starts_original + NFFT < (int)ts_size];
    /* create a 2D vector where rows represent each time window and columns represent frequency bins */
    vector<vector<float>> xns (starts.size(), vector<float>(ts_size/2));

    nc::NdArray<int> ks = nc::arange<int>(0, NFFT, 1);
    printf("%ld computations will occur\n", starts.size() * (ts_size/2) * NFFT);
    auto start = high_resolution_clock::now();
    for (int m = 0; m < starts.size(); m++) {

        dsp::DFT_slow(ts, &ks, &(xns[m]), starts[m], NFFT);

        // for (int n = 0; n < ts_size/2; n++) {
        //     dcomp a = 0;
        //     float calc = 0.0;
        //     int ts_offset = starts[m];

        //     dcomp curr_n = n;
        //     dcomp curr_NFFT = NFFT;
        //     dcomp curr_pi = M_PI;
        //     dcomp two = 2;

        //     for (int k = 0; k < NFFT; k++) {
        //         dcomp curr_ts = (*ts)[ts_offset + k];
        //         dcomp curr_ks = ks[k];
        //         a += curr_ts * exp((img * two * curr_pi * curr_ks * curr_n)/curr_NFFT);
        //     }

        //     calc = 10*log10(abs(a)*2);

        //     xns[m][n] = calc;
        // }
    }
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Execution time: " << duration.count() << endl;
    /* print out the dft values */
    
    for (int m = 0; m < 2; m++) {
        for (int n = 0; n < ts_size / 2; n++) {
            printf("%.3f ", xns[m][n]);
        }
        printf("\n");
    }

    return 0;
}

__host__ int dsp::DFT_slow(vector<float> *ts, nc::NdArray<int> *ks, vector<float> *xns, int ts_offset, int NFFT) {
    int ts_size = ts->size();
    for (int n = 0; n < ts_size/2; n++) {
            dcomp a = 0;
            float calc = 0.0;

            dcomp curr_n = n;
            dcomp curr_NFFT = NFFT;
            dcomp curr_pi = M_PI;
            dcomp two = 2;

            for (int k = 0; k < NFFT; k++) {
                dcomp curr_ts = (*ts)[ts_offset + k];
                dcomp curr_ks = (*ks)[k];
                a += curr_ts * exp((img * two * curr_pi * curr_ks * curr_n)/curr_NFFT);
            }

            calc = 10*log10(abs(a)*2);

            (*xns)[n] = calc;
    }
    return -1;
}

/* this will be the serial CPU FFT version which should display speed up over DFT_slow */
__host__ void dsp::FFT(const float* samples, complex<double>* freqs, const int num_samples) {

    unsigned char idx_arr[4];
    unsigned int input_idx;
    int bit_shift = (int)log2((float)num_samples); // also corresponds to number of stages 
    int sw = 0;
    // complex<double> shmem [num_samples * 2.5];
    complex<double>* shmem = (complex<double>*)malloc(num_samples * 2.5 * sizeof(complex<double>));

    #define in(i0, swi) shmem[swi*num_samples + i0]
    #define twiddle(i0) shmem[2*num_samples + i0]

    for (int tx = 0; tx < num_samples; tx++) {

        /* rearrange smaples into necessary order for FFT */
        for(int i = 0; i < 4; i++)
            idx_arr[i] = dsp::reverse_table[(0x000000FF) & (tx >> (i*8))];

        input_idx = (unsigned int)(idx_arr[0] << 24 | idx_arr[1] << 16 | idx_arr[2] << 8 | idx_arr[3]);
        input_idx = input_idx >> (32 - bit_shift);
        
        /* copy inputs to shared memory */
        if (tx < num_samples)
            in(input_idx, sw) = complex<double>(samples[tx], 0.0); 

        /* only need half the twiddle factors since they are symmetric */
        if (tx < num_samples/2)
            twiddle(tx) = exp((complex<double>(0.0, -2.0) * complex<double>(M_PI, 0.0) * complex<double>(tx, 0.0)) /  complex<double>(num_samples, 0.0));

    }

    /* perform FFT in stages */
    int gs = 2; // the size of each DFT being computed, thus N/gs is the number of groups
    int gs_idx; // idx of thread in the group
    int twiddle_idx; // idx of twiddle factor
    int pair_tx; // the thread idx that the current thread must share data with
    
    for (int i = 0; i < bit_shift; i++) {
        for (int tx = 0; tx < num_samples; tx++) {
                gs_idx = tx % gs;
                /* this is the positive member of the pair*/
                /* NOTE: this will cause divergence, try and see if there is a way to prevent this */
                if ( (float)gs_idx < (1.0*gs/2) ) {
                    pair_tx = tx + (gs/2);
                    twiddle_idx = (int)(((float)gs_idx / gs)*num_samples);
                    in(tx, !sw) = in(tx, sw) + twiddle(twiddle_idx) * in(pair_tx, sw);
                }
                /* negative member */
                else {
                    pair_tx = tx - (gs/2);
                    twiddle_idx = (int)((((float)gs_idx - gs/2) / gs)*num_samples);
                    in(tx, !sw) = in(pair_tx, sw) - twiddle(twiddle_idx) * in(tx, sw);
                }
            }
        gs *= 2; // number of elements in a group will double
        sw = !sw;
        }

    for (int tx = 0; tx < num_samples; tx++)
        freqs[tx] = in(tx, sw);

    free(shmem);
    
    return;

    #undef in
    #undef twiddle
}

// #define N 10000000
// #define REVERSE_TABLE_SIZE 256

// __constant__ unsigned char device_reverse_table[REVERSE_TABLE_SIZE];

__global__ void dsp::vector_add(float *out, float *a, float *b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    	out[idx] = a[idx] + b[idx];
}

/* calculates the complex float exponent */
__device__ __forceinline__ hipFloatComplex my_cexpf (hipFloatComplex z) {
    hipFloatComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ __forceinline__ hipDoubleComplex my_cexp (hipDoubleComplex z) {
    hipDoubleComplex res;
    double t = exp (z.x);
    sincos (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__host__ int dsp::cuFFT(float* samples, hipDoubleComplex* freqs, int num_samples) {

    /* create device pointers */
    float* device_samples;
    hipDoubleComplex* device_freqs;

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, num_samples*sizeof(hipDoubleComplex)));
    size_t shmemsize = num_samples * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reverse_table), dsp::reverse_table, REVERSE_TABLE_SIZE*sizeof(unsigned char)));
    gpuErrchk(hipMemcpy(device_samples, samples, num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    dim3 blockDim(maxThreads > num_samples ? num_samples : maxThreads, 1, 1);
    dim3 gridDim(ceil((float)num_samples / maxThreads), 1, 1);

    /* kernel invocation */
    FFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, num_samples);

    /* synchronize and copy data back to host */
    hipDeviceSynchronize();
    hipMemcpy(freqs, device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    /* free memory */
    hipFree(device_samples);
    hipFree(device_freqs);

    return maxThreads;
}

/* python function to be called */
// __host__ vector<complex<double>> dsp::pybind_cuFFT(vector<float> samples, vector<complex<double>> freqs) {

//     /* create device pointers */
//     float* device_samples;
//     hipDoubleComplex* device_freqs;
//     int num_samples = samples.size();

//     /* allocate memory for device and shared memory */
//     gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
//     gpuErrchk(hipMalloc((void**)&device_freqs, num_samples*sizeof(hipDoubleComplex)));
//     size_t shmemsize = num_samples * 2.5 * sizeof(hipDoubleComplex);

//     /* copy data to device and constant memory */
//     gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reverse_table), dsp::reverse_table, REVERSE_TABLE_SIZE*sizeof(unsigned char)));
//     gpuErrchk(hipMemcpy(device_samples, &samples[0], num_samples*sizeof(float), hipMemcpyHostToDevice));

//     /* get max threads per block and create dimensions */
//     int maxThreads = dsp::get_thread_per_block();

//     dim3 blockDim(maxThreads > num_samples ? num_samples : maxThreads, 1, 1);
//     dim3 gridDim(ceil((float)num_samples / maxThreads), 1, 1);

//     /* kernel invocation */
//     FFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, num_samples);

//     /* synchronize and copy data back to host */
//     hipDeviceSynchronize();
//     hipMemcpy(&freqs[0], device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

//     /* free memory */
//     hipFree(device_samples);
//     hipFree(device_freqs);

//     return freqs;
// }

/* note that the max FFT size is limited to the max number of threads allowed in a thread block */
__global__ void dsp::FFT_Kernel(const float* samples, hipDoubleComplex* __restrict__ freqs, const int num_samples) {
    int tx = threadIdx.x;
    unsigned char idx_arr[4]; // character array used to create input_idx
    unsigned int input_idx; // sample index each thread is responsible for loading to shared memory 
    int bit_shift = (int)log2f((float)num_samples); // also corresponds to number of stages 
    int sw = 0; // flag for alternating computational buffers 
    extern __shared__ hipDoubleComplex shmem []; // will be used to hold the inputs/computations and 'twiddle' factors

    /* defines for simpler access to shared memory */
    #define in(i0, swi) shmem[swi*num_samples + i0]
    #define twiddle(i0) shmem[2*num_samples + i0]

    /* rearrange samples into necessary order in shared memory */
    for(int i = 0; i < 4; i++)
        idx_arr[i] = device_reverse_table[(0x000000FF) & (tx >> (i*8))];

    input_idx = (unsigned int)(idx_arr[0] << 24 | idx_arr[1] << 16 | idx_arr[2] << 8 | idx_arr[3]);
    input_idx = input_idx >> (32 - bit_shift);
    
    /* copy inputs to shared memory */
    if (tx < num_samples)
        in(input_idx, sw) = make_hipDoubleComplex(samples[tx], 0.0); 

    /* only need half the twiddle factors since they are symmetric */
    if (tx < num_samples/2)
        twiddle(tx) = my_cexp(hipCdiv(hipCmul(hipCmul(make_hipDoubleComplex(0.0, -2.0), make_hipDoubleComplex(M_PI, 0.0)), make_hipDoubleComplex(tx, 0.0)), make_hipDoubleComplex(num_samples, 0.0)));

    /* perform FFT in stages */
    int gs = 2; // the size of each DFT being computed, thus N/gs is the number of groups
    int gs_idx; // idx of thread in the group
    int twiddle_idx; // idx of twiddle factor
    int pair_tx; // the thread idx that the current thread must share data with
    
    if (tx < num_samples) {
        for (int i = 0; i < bit_shift; i++) {
            __syncthreads();
            gs_idx = tx % gs;
            /* this is the positive member of the pair*/
            /* NOTE: this will cause divergence, try and see if there is a way to prevent this */
            if ( (float)gs_idx < (1.0*gs/2) ) {
                pair_tx = tx + (gs/2);
                twiddle_idx = (int)(((float)gs_idx / gs)*num_samples);
                in(tx, !sw) = hipCadd(in(tx, sw), hipCmul(twiddle(twiddle_idx),in(pair_tx, sw)));
            }
            /* negative member */
            else {
                pair_tx = tx - (gs/2);
                twiddle_idx = (int)((((float)gs_idx - gs/2) / gs)*num_samples);
                in(tx, !sw) = hipCsub(in(pair_tx, sw), hipCmul(twiddle(twiddle_idx),in(tx, sw)));
            }
            gs *= 2; // number of elements in a group will double
            sw = !sw;
        }
    }

    __syncthreads();

    /* return the magnitude as the final output */
    if (tx < num_samples) 
        freqs[tx] = in(tx, sw);

    #undef in
    #undef twiddle
}

__host__ int dsp::cuSTFT(float* samples, hipDoubleComplex** freqs, int num_samples, int NFFT, int noverlap = -1) {

    /* default noverlap */
    if (noverlap < 0)
        noverlap = NFFT / 2;

    int step = NFFT - noverlap;
    int num_ffts = ceil((float)num_samples/step);

    /* trim FFT's that are out of bounds */
    while ( num_ffts * step >= num_samples )
        num_ffts--;

    int xns_size = num_ffts * NFFT;
    hipDoubleComplex* xns = (hipDoubleComplex*)malloc(xns_size*sizeof(hipDoubleComplex));
    mallocErrchk(xns);

    /* create device pointers */
    float* device_samples;
    hipDoubleComplex* device_freqs;

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, xns_size*sizeof(hipDoubleComplex)));
    size_t shmemsize = NFFT * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reverse_table), dsp::reverse_table, REVERSE_TABLE_SIZE*sizeof(unsigned char)));
    gpuErrchk(hipMemcpy(device_samples, samples, num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    // Set dimensions
    dim3 blockDim(maxThreads > NFFT ? NFFT : maxThreads, 1, 1);
    dim3 gridDim(num_ffts, 1, 1);

    printf("block dim: x.%d, y.%d, z.%d\n", blockDim.x, blockDim.y, blockDim.z);
    printf("grid dim: x.%d, y.%d, z.%d\n", gridDim.x, gridDim.y, gridDim.z);

    /* kernel invocation */
    dsp::STFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, NFFT, step);

    /* synchronize and copy data back to host */
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpy(xns, device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    
    /* free memory */
    gpuErrchk(hipFree(device_samples));
    gpuErrchk(hipFree(device_freqs));

    *freqs = xns;
   
    return xns_size;
}

/* note that the max FFT size is limited to the max number of threads allowed in a thread block */
__global__ void dsp::STFT_Kernel(const float* samples, hipDoubleComplex* __restrict__ freqs, const int num_samples, int step) {
    // NOTE; here num_samples is equivalent to NFFT not that actual number of total samples
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    unsigned char idx_arr[4]; // character array used to create input_idx
    unsigned int input_idx; // sample index each thread is responsible for loading to shared memory 
    int bit_shift = (int)log2f((float)num_samples); // also corresponds to number of stages 
    int sw = 0; // flag for alternating computational buffers 
    extern __shared__ hipDoubleComplex shmem []; // will be used to hold the inputs/computations and 'twiddle' factors

    /* defines for simpler access to shared memory */
    #define in(i0, swi) shmem[swi*num_samples + i0]
    #define twiddle(i0) shmem[2*num_samples + i0]

    /* rearrange samples into necessary order in shared memory */
    for(int i = 0; i < 4; i++)
        idx_arr[i] = device_reverse_table[(0x000000FF) & (tx >> (i*8))];

    input_idx = (unsigned int)(idx_arr[0] << 24 | idx_arr[1] << 16 | idx_arr[2] << 8 | idx_arr[3]);
    input_idx = input_idx >> (32 - bit_shift);
    
    /* copy inputs to shared memory */
    if (tx < num_samples)
        in(input_idx, sw) = make_hipDoubleComplex(samples[tx + bx*step], 0.0); 

    /* only need half the twiddle factors since they are symmetric */
    if (tx < num_samples/2)
        twiddle(tx) = my_cexp(hipCdiv(hipCmul(hipCmul(make_hipDoubleComplex(0.0, -2.0), make_hipDoubleComplex(M_PI, 0.0)), make_hipDoubleComplex(tx, 0.0)), make_hipDoubleComplex(num_samples, 0.0)));

    /* perform FFT in stages */
    int gs = 2; // the size of each DFT being computed, thus N/gs is the number of groups
    int gs_idx; // idx of thread in the group
    int twiddle_idx; // idx of twiddle factor
    int pair_tx; // the thread idx that the current thread must share data with
    
    if (tx < num_samples) {
        for (int i = 0; i < bit_shift; i++) {
            __syncthreads();
            gs_idx = tx % gs;
            /* this is the positive member of the pair*/
            /* NOTE: this will cause divergence, try and see if there is a way to prevent this */
            if ( (float)gs_idx < (1.0*gs/2) ) {
                pair_tx = tx + (gs/2);
                twiddle_idx = (int)(((float)gs_idx / gs)*num_samples);
                in(tx, !sw) = hipCadd(in(tx, sw), hipCmul(twiddle(twiddle_idx),in(pair_tx, sw)));
            }
            /* negative member */
            else {
                pair_tx = tx - (gs/2);
                twiddle_idx = (int)((((float)gs_idx - gs/2) / gs)*num_samples);
                in(tx, !sw) = hipCsub(in(pair_tx, sw), hipCmul(twiddle(twiddle_idx),in(tx, sw)));
            }
            gs *= 2; // number of elements in a group will double
            sw = !sw;
        }
    }

    __syncthreads();

    /* return the magnitude as the final output */
    if (tx < num_samples) 
        freqs[tx + bx*num_samples] = in(tx, sw);

    #undef in
    #undef twiddle
}

__host__ void dsp::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
        std::cout<<"Size of hipDoubleComplex: "<<sizeof(hipDoubleComplex)<<endl;
    }
}

/* assumes only one device is being utilized */
__host__ int dsp::get_thread_per_block() {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    return deviceProp.maxThreadsPerBlock;
}

// __host__ int dsp::test_cuda(){
//     float *a, *b, *out;
//     float *d_a, *d_b, *d_out;

//     a = (float*)malloc(sizeof(float) * N);
//     b = (float*)malloc(sizeof(float) * N);
//     out = (float*)malloc(sizeof(float) * N);
//     for (int i = 0; i < N; i++) {
// 	    *(a + i) = 1.0;
// 	    *(b + i) = 1.0;
//     }

//     // Allocate device memory for a
//     hipMalloc((void**)&d_a, sizeof(float) * N);
//     hipMalloc((void**)&d_b, sizeof(float) *N);
//     hipMalloc((void**)&d_out, sizeof(float) *N);

//     // Transfer data from host to device memory
//     hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
//     hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
//     dim3 gridDim(ceil(1.0*N/1024), 1, 1);
//     dim3 blockDim(1024, 1, 1);
//     vector_add<<<gridDim,blockDim>>>(d_out, d_a, d_b, N);
//     hipDeviceSynchronize();
//     hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
//     for (int i = 0; i < 10; i++)
// 	    printf("%f ", *(out + i));
   
//     get_device_properties();
//     // Cleanup after kernel execution
//     hipFree(d_a);
//     hipFree(d_b);
//     hipFree(d_out);

//     free(a);
//     free(b);
//     free(out);

//     return 1;
// }

// int test_function () {return 1;}

// PYBIND11_MODULE(dsp_module, module_handle) {
//     module_handle.doc() = "I'm a docstring hehe";
//     module_handle.def("get_thread_per_block", &dsp::get_thread_per_block);
// //   module_handle.def("cuFFT", &dsp::pybind_cuFFT);
//     module_handle.def("test_func", &test_function);
// //   module_handle.def("some_fn_python_name", &some_fn);
// //   module_handle.def("some_class_factory", &some_class_factory);
// //   py::class_<SomeClass>(
// // 			module_handle, "PySomeClass"
// // 			).def(py::init<float>())
// //     .def_property("multiplier", &SomeClass::get_mult, &SomeClass::set_mult)
// //     .def("multiply", &SomeClass::multiply)
// //     .def("multiply_list", &SomeClass::multiply_list)
// //     // .def_property_readonly("image", &SomeClass::make_image)
// //     .def_property_readonly("image", [](SomeClass &self) {
// // 				      py::array out = py::cast(self.make_image());
// // 				      return out;
// // 				    })
// //     // .def("multiply_two", &SomeClass::multiply_two)
// //     .def("multiply_two", [](SomeClass &self, float one, float two) {
// // 			   return py::make_tuple(self.multiply(one), self.multiply(two));
// // 			 })
// //     .def("function_that_takes_a_while", &SomeClass::function_that_takes_a_while)
// //     ;
// }