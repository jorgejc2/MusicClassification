#include "hip/hip_runtime.h"
#include "dsp_pybind/dsp.pybind.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <iostream>

namespace py = pybind11;

/* checks for CUDA errors */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* checks if memory could not be allocated */
#define mallocErrchk(ans) { mallocAssert((ans), __FILE__, __LINE__); }
inline void mallocAssert(void* pointer, const char *file, int line, bool abort=true) {
    if (pointer == nullptr)
    {
        fprintf(stderr, "mallocAssert: Returns nullptr at %s %d\n", file, line);
        if (abort) exit(1);
    }
}

/* calculates the complex float exponent */
__device__ __forceinline__ hipFloatComplex my_cexpf (hipFloatComplex z) {
    hipFloatComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ __forceinline__ hipDoubleComplex my_cexp (hipDoubleComplex z) {
    hipDoubleComplex res;
    double t = exp (z.x);
    sincos (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

int test_function () {return 1;}

__host__ int test_cuda(){
    int N = 10000000;

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; i++) {
	    *(a + i) = 4.0;
	    *(b + i) = 1.0;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) *N);
    hipMalloc((void**)&d_out, sizeof(float) *N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.0*N/1024), 1, 1);
    dim3 blockDim(1024, 1, 1);
    dsp::vector_add<<<gridDim,blockDim>>>(d_out, d_a, d_b, N);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
	    printf("%f ", *(out + i));
   
    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    return 1;
}

// __constant__ unsigned char device_reverse_table[REVERSE_TABLE_SIZE];

__host__ vector<complex<double>> pybind_cuFFT(vector<float> samples) {
    /* NOTE: complex<double> on host seems to cast well with hipDoubleComplex but not sure if always true */
    int num_samples = samples.size();

    /* create device pointers */
    float* device_samples;
    hipDoubleComplex* device_freqs;

    /* initialize empty freqs vector to return */
    vector<complex<double>> freqs(num_samples, complex<double>(0,0));

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, num_samples*sizeof(hipDoubleComplex)));
    size_t shmemsize = num_samples * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    dsp::cpy_to_symbol();
    gpuErrchk(hipMemcpy(device_samples, &samples[0], num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    dim3 blockDim(maxThreads > num_samples ? num_samples : maxThreads, 1, 1);
    dim3 gridDim(ceil((float)num_samples / maxThreads), 1, 1);

    /* kernel invocation */
    dsp::FFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, num_samples);

    /* synchronize and copy data back to host */
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk(hipMemcpy(&freqs[0], device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    /* free memory */
    gpuErrchk(hipFree(device_samples));
    gpuErrchk(hipFree(device_freqs));

    return freqs;
}

PYBIND11_MODULE(dsp_module, module_handle) {
    module_handle.doc() = "I'm a docstring hehe";
    module_handle.def("get_thread_per_block", &dsp::get_thread_per_block);
    module_handle.def("cuFFT", &pybind_cuFFT, py::return_value_policy::copy);
    module_handle.def("test_func", &test_function);
    module_handle.def("test_cuda", &test_cuda);
//   module_handle.def("some_fn_python_name", &some_fn);
//   module_handle.def("some_class_factory", &some_class_factory);
//   py::class_<SomeClass>(
// 			module_handle, "PySomeClass"
// 			).def(py::init<float>())
//     .def_property("multiplier", &SomeClass::get_mult, &SomeClass::set_mult)
//     .def("multiply", &SomeClass::multiply)
//     .def("multiply_list", &SomeClass::multiply_list)
//     // .def_property_readonly("image", &SomeClass::make_image)
//     .def_property_readonly("image", [](SomeClass &self) {
// 				      py::array out = py::cast(self.make_image());
// 				      return out;
// 				    })
//     // .def("multiply_two", &SomeClass::multiply_two)
//     .def("multiply_two", [](SomeClass &self, float one, float two) {
// 			   return py::make_tuple(self.multiply(one), self.multiply(two));
// 			 })
//     .def("function_that_takes_a_while", &SomeClass::function_that_takes_a_while)
//     ;
}