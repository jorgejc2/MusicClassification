#include "hip/hip_runtime.h"
#include "dsp_pybind/dsp.pybind.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <iostream>

namespace py = pybind11;

/* checks for CUDA errors */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* checks if memory could not be allocated */
#define mallocErrchk(ans) { mallocAssert((ans), __FILE__, __LINE__); }
inline void mallocAssert(void* pointer, const char *file, int line, bool abort=true) {
    if (pointer == nullptr)
    {
        fprintf(stderr, "mallocAssert: Returns nullptr at %s %d\n", file, line);
        if (abort) exit(1);
    }
}

/* calculates the complex float exponent */
__device__ __forceinline__ hipFloatComplex my_cexpf (hipFloatComplex z) {
    hipFloatComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ __forceinline__ hipDoubleComplex my_cexp (hipDoubleComplex z) {
    hipDoubleComplex res;
    double t = exp (z.x);
    sincos (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

int test_function () {return 1;}

__host__ int test_cuda(){
    int N = 10000000;

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; i++) {
	    *(a + i) = 4.0;
	    *(b + i) = 1.0;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) *N);
    hipMalloc((void**)&d_out, sizeof(float) *N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.0*N/1024), 1, 1);
    dim3 blockDim(1024, 1, 1);
    dsp::vector_add<<<gridDim,blockDim>>>(d_out, d_a, d_b, N);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
	    printf("%f ", *(out + i));
   
    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    return 1;
}

// __constant__ unsigned char device_reverse_table[REVERSE_TABLE_SIZE];

__host__ vector<complex<double>> pybind_cuFFT(vector<float> samples) {
    int num_samples = samples.size();
    // printf("Numsamples: %d\n", num_samples);

    /* create device pointers */
    float* device_samples;
    // hipDoubleComplex cu_freqs [num_samples];
    float* init_samples = (float*)malloc(num_samples * sizeof(float));
    for (int i = 0; i < num_samples; i++)
        init_samples[i] = samples[i];
    hipDoubleComplex* cu_freqs = (hipDoubleComplex*)malloc(num_samples * sizeof(hipDoubleComplex));
    hipDoubleComplex* device_freqs;
    

    // printf("First three samples\n");
    // for(int i = 0; i < 3; i++)
    //     printf("%f\n", init_samples[i]);

    /* initialize empty freqs vector to return */
    vector<complex<double>> freqs(num_samples, complex<double>(0,0));

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, num_samples*sizeof(hipDoubleComplex)));
    size_t shmemsize = num_samples * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reverse_table), dsp::reverse_table, REVERSE_TABLE_SIZE*sizeof(unsigned char)));
    dsp::cpy_to_symbol();
    gpuErrchk(hipMemcpy(device_samples, init_samples, num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    dim3 blockDim(maxThreads > num_samples ? num_samples : maxThreads, 1, 1);
    dim3 gridDim(ceil((float)num_samples / maxThreads), 1, 1);

    // printf("maxThreads: %d, blockDim.x: %d, gridDim.x: %d, shmemsize: %ld\n", maxThreads, blockDim.x, gridDim.x, shmemsize);

    /* kernel invocation */
    dsp::FFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, num_samples);

    /* synchronize and copy data back to host */
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk(hipMemcpy(cu_freqs, device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    // for (int i = 0; i < num_samples; i++)
    //     freqs[i] = complex<double>(cu_freqs[i].x, cu_freqs[i].y);
    // printf("First three freqs\n");
    // for(int i = 0; i < 20; i++)
    //     printf("%ld %ld\n", real(freqs[i]), imag(freqs[i]));

    /* free memory */
    gpuErrchk(hipFree(device_samples));
    gpuErrchk(hipFree(device_freqs));

    for (int i = 0; i < num_samples; i++) {
        freqs[i] = complex<double>(cu_freqs[i].x, cu_freqs[i].y);
    }
        
    // printf("First three freqs\n");
    // for(int i = 0; i < 20; i++)
    //     cout<<freqs[i]<<endl;
    //     // printf("%ld %ld\n", real(freqs[i]), imag(freqs[i]));

    // vector<complex<double>> test = {
    //     complex<double>(1,1),
    //     complex<double>(2,2),
    //     complex<double>(3,3)
    // };
    free(init_samples);
    free(cu_freqs);
    return freqs;
}

PYBIND11_MODULE(dsp_module, module_handle) {
    module_handle.doc() = "I'm a docstring hehe";
    module_handle.def("get_thread_per_block", &dsp::get_thread_per_block);
    module_handle.def("cuFFT", &pybind_cuFFT, py::return_value_policy::copy);
    module_handle.def("test_func", &test_function);
    module_handle.def("test_cuda", &test_cuda);
//   module_handle.def("some_fn_python_name", &some_fn);
//   module_handle.def("some_class_factory", &some_class_factory);
//   py::class_<SomeClass>(
// 			module_handle, "PySomeClass"
// 			).def(py::init<float>())
//     .def_property("multiplier", &SomeClass::get_mult, &SomeClass::set_mult)
//     .def("multiply", &SomeClass::multiply)
//     .def("multiply_list", &SomeClass::multiply_list)
//     // .def_property_readonly("image", &SomeClass::make_image)
//     .def_property_readonly("image", [](SomeClass &self) {
// 				      py::array out = py::cast(self.make_image());
// 				      return out;
// 				    })
//     // .def("multiply_two", &SomeClass::multiply_two)
//     .def("multiply_two", [](SomeClass &self, float one, float two) {
// 			   return py::make_tuple(self.multiply(one), self.multiply(two));
// 			 })
//     .def("function_that_takes_a_while", &SomeClass::function_that_takes_a_while)
//     ;
}