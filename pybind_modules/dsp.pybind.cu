#include "hip/hip_runtime.h"
#include "dsp_pybind/dsp.pybind.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <iostream>

namespace py = pybind11;

/* checks for CUDA errors */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* checks if memory could not be allocated */
#define mallocErrchk(ans) { mallocAssert((ans), __FILE__, __LINE__); }
inline void mallocAssert(void* pointer, const char *file, int line, bool abort=true) {
    if (pointer == nullptr)
    {
        fprintf(stderr, "mallocAssert: Returns nullptr at %s %d\n", file, line);
        if (abort) exit(1);
    }
}

/* calculates the complex float exponent */
__device__ __forceinline__ hipFloatComplex my_cexpf (hipFloatComplex z) {
    hipFloatComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ __forceinline__ hipDoubleComplex my_cexp (hipDoubleComplex z) {
    hipDoubleComplex res;
    double t = exp (z.x);
    sincos (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

int test_function () {return 1;}

__host__ int test_cuda(){
    int N = 10000000;

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; i++) {
	    *(a + i) = 4.0;
	    *(b + i) = 1.0;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) *N);
    hipMalloc((void**)&d_out, sizeof(float) *N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.0*N/1024), 1, 1);
    dim3 blockDim(1024, 1, 1);
    dsp::vector_add<<<gridDim,blockDim>>>(d_out, d_a, d_b, N);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
	    printf("%f ", *(out + i));
   
    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    return 1;
}

__host__ vector<complex<double>> pybind_cuFFT(vector<float> samples) {
    /* NOTE: complex<double> on host seems to cast well with hipDoubleComplex but not sure if always true */
    int num_samples = samples.size();

    /* create device pointers */
    float* device_samples;
    hipDoubleComplex* device_freqs;

    /* initialize empty freqs vector to return */
    vector<complex<double>> freqs(num_samples, complex<double>(0,0));

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, num_samples*sizeof(hipDoubleComplex)));
    size_t shmemsize = num_samples * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    dsp::cpy_to_symbol();
    gpuErrchk(hipMemcpy(device_samples, &samples[0], num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    dim3 blockDim(maxThreads > num_samples ? num_samples : maxThreads, 1, 1);
    dim3 gridDim(ceil((float)num_samples / maxThreads), 1, 1);

    /* kernel invocation */
    dsp::FFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, num_samples);

    /* synchronize and copy data back to host */
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk(hipMemcpy(&freqs[0], device_freqs, num_samples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    /* free memory */
    gpuErrchk(hipFree(device_samples));
    gpuErrchk(hipFree(device_freqs));

    return freqs;
}

__host__ vector<vector<complex<double>>> pybind_cuSTFT(vector<float> samples, int NFFT, int noverlap) {

    /* initialization */
    int num_samples = samples.size(); // get number of samples

    /* default noverlap */
    if (noverlap < 0)
        noverlap = NFFT / 2;

    int step = NFFT - noverlap;
    int num_ffts = ceil((float)num_samples/step);

    /* trim FFT's that are out of bounds */
    while ( num_ffts * step >= num_samples )
        num_ffts--;

    int xns_size = num_ffts * NFFT;
    vector<vector<complex<double>>> xns(NFFT, vector<complex<double>>(num_ffts, complex<double>(0,0)));
    hipDoubleComplex* freqs = (hipDoubleComplex*)malloc(xns_size*sizeof(hipDoubleComplex));
    mallocErrchk(freqs);
    /* create device pointers */
    float* device_samples;
    hipDoubleComplex* device_freqs;

    /* allocate memory for device and shared memory */
    gpuErrchk(hipMalloc((void**)&device_samples, num_samples*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&device_freqs, xns_size*sizeof(hipDoubleComplex)));
    size_t shmemsize = NFFT * 2.5 * sizeof(hipDoubleComplex);

    /* copy data to device and constant memory */
    dsp::cpy_to_symbol();
    gpuErrchk(hipMemcpy(device_samples, &samples[0], num_samples*sizeof(float), hipMemcpyHostToDevice));

    /* get max threads per block and create dimensions */
    int maxThreads = dsp::get_thread_per_block();

    // Set dimensions
    dim3 blockDim(maxThreads > NFFT ? NFFT : maxThreads, 1, 1);
    dim3 gridDim(num_ffts, 1, 1);

    // printf("block dim: x.%d, y.%d, z.%d\n", blockDim.x, blockDim.y, blockDim.z);
    // printf("grid dim: x.%d, y.%d, z.%d\n", gridDim.x, gridDim.y, gridDim.z);

    /* kernel invocation */
    dsp::STFT_Kernel<<<gridDim, blockDim, shmemsize>>>(device_samples, device_freqs, NFFT, step);

    /* synchronize and copy data back to host */
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    /* may be an issue copying array into a 2D vector */
    gpuErrchk(hipMemcpy(freqs, device_freqs, xns_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    
    /* free memory */
    gpuErrchk(hipFree(device_samples));
    gpuErrchk(hipFree(device_freqs));

    for (int i = 0; i < NFFT; i++) {
        for (int j = 0; j < num_ffts; j++) {
            xns[i][j] = complex<double>(freqs[i * num_ffts + j].x, freqs[i * num_ffts + j].y);
        }
    }

    free(freqs);

    // for (int i = 0; i < 8; i++) {
    //     printf("%f\n", xns[0][i]);
    // }
   
    return xns;
}

PYBIND11_MODULE(dsp_module, module_handle) {
    module_handle.doc() = "I'm a docstring hehe";
    module_handle.def("get_thread_per_block", &dsp::get_thread_per_block);
    module_handle.def("cuFFT", &pybind_cuFFT, py::return_value_policy::copy);
    // module_handle.def("cuSTFT", &pybind_cuSTFT, py::return_value_policy::copy);
    module_handle.def("cuSTFT", [](vector<float> samples, int NFFT, int noverlap) {
        printf("len(samples): %d, NFFT: %d, noverlap: %d\n", samples.size(), NFFT, noverlap);
        py::array out = py::cast(pybind_cuSTFT(samples, NFFT, noverlap));
        return out;
    }, py::arg("samples"), py::arg("NFFT"), py::arg("noverlap"), py::return_value_policy::move);
    module_handle.def("test_func", &test_function);
    module_handle.def("test_cuda", &test_cuda);
/* commented out but kept for reference for adding a class */

//   module_handle.def("some_fn_python_name", &some_fn);
//   module_handle.def("some_class_factory", &some_class_factory);
//   py::class_<SomeClass>(
// 			module_handle, "PySomeClass"
// 			).def(py::init<float>())
//     .def_property("multiplier", &SomeClass::get_mult, &SomeClass::set_mult)
//     .def("multiply", &SomeClass::multiply)
//     .def("multiply_list", &SomeClass::multiply_list)
//     // .def_property_readonly("image", &SomeClass::make_image)
//     .def_property_readonly("image", [](SomeClass &self) {
// 				      py::array out = py::cast(self.make_image());
// 				      return out;
// 				    })
//     // .def("multiply_two", &SomeClass::multiply_two)
//     .def("multiply_two", [](SomeClass &self, float one, float two) {
// 			   return py::make_tuple(self.multiply(one), self.multiply(two));
// 			 })
//     .def("function_that_takes_a_while", &SomeClass::function_that_takes_a_while)
//     ;
}